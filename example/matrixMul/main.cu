#include "hip/hip_runtime.h"
// C++ imports
#include <iostream>
#include<cstdio>
#include <ctime>
#include <cmath>
#include "bits/time.h"

//#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include "opencv2/core/cuda.hpp"
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/imgcodecs.hpp>

#include<hip/hip_runtime.h>

// namespaces
using namespace std;
using namespace cv;
#define PI 3.1415926
#define DIM 4

// int frameWidth = 640;
// int frameHeight = 480;
#define FRAMEWIDTH  640
#define FRAMEHEIGHT 480



void stampaMatrice(float *matrice, int rig, int col){
    int idx;  
    //stampa a matrice
    for(int i = 0; i < rig; i++){
        for(int j = 0; j < col; j++){
            if (i == 0){
                idx = j;
            }else{
                idx = i * col + j;
            }
            cout << matrice[idx] << "\t";
        }
        cout<<"\n";
    }
    cout<<"\n\n";
    
}

/**

*/
__global__ void generic_mat_mul(float *A, float *B, float *C, int numARows,int numAColumns, int numBRows, int numBColumns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numARows && col < numBColumns) {
        float sum = 0;
        for (int ii = 0; ii < numAColumns; ii++) {
            sum += A[row * numAColumns + ii] * B[ii * numBColumns + col];
        }
        C[row * numBColumns + col] = sum;
    }
}

/*
__global__ void square_mat_mul(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    if (ROW < N && COL < N) {
        float tmpSum = 0;
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
        C[ROW * N + COL] = tmpSum;
    }
    
}
*/

/**
    A * B = C
    N = numero di colonne
*/

hipError_t matrixMultiplication(float *A, float *B, float *C, int numARows,int numAColumns, int numBRows, int numBColumns){
    hipError_t cudaStatus;
    //@@ Initialize the grid and block dimensions here
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((float)numAColumns) / blockDim.x),ceil(((float)numBRows) / blockDim.y));
    float *d_A, *d_B, *d_C;

    cudaStatus = hipMalloc((void **) &d_A, sizeof(float)*numARows*numAColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void **) &d_B, sizeof(float)*numBRows*numBColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void **) &d_C, sizeof(float)*numARows * numBColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    //copio i vettori
    hipMemcpy(d_A,A,sizeof(float)*numARows*numAColumns,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,sizeof(float)*numBRows*numBColumns,hipMemcpyHostToDevice);
    
   

    hipMemset(d_C, 0, numARows * numBColumns * sizeof(float));

    generic_mat_mul<<<gridDim, blockDim>>>(d_A, d_B, d_C, 2, 2, 2, 2);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(C, d_C,numARows * numBColumns * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    //@@ Free the GPU memory here
Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return cudaStatus;
    
}


int main(int argc, const char *argv[]) {
    //float RX[16] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 1, 2, 3, 4, 5, 6, 7};

    float RX[DIM] = { 1, 2, 3, 4};
    float RY[DIM] = { 1, 2, 3, 4};
    
    float ris[DIM];
    cout << "ciao!\n";
    // matrixMultiplication(float *A, float *B, float *C, int numARows,int numAColumns, int numBRows, int numBColumns){

    hipError_t cudaStatus = matrixMultiplication(RX,RY,ris, 2, 2, 2, 2);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    stampaMatrice(ris, 2, 2);

}