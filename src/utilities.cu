#include "hip/hip_runtime.h"
#include "../lib/utilities.h"

using namespace std;
using namespace cv;

void stampaMatrice(float *matrice, int rig, int col){
    int idx;  
    //stampa a matrice
    for(int i = 0; i < rig; i++){
        for(int j = 0; j < col; j++){
            if (i == 0){
                idx = j;
            }else{
                idx = i * col + j;
            }
            cout << matrice[idx] << "\t";
        }
        cout<<"\n";
    }
    cout<<"\n\n";
    
}

/**

*/
__global__ void generic_mat_mul(float *A, float *B, float *C, int numARows,int numAColumns, int numBRows, int numBColumns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numARows && col < numBColumns) {
        float sum = 0;
        for (int ii = 0; ii < numAColumns; ii++) {
            sum += A[row * numAColumns + ii] * B[ii * numBColumns + col];
        }
        C[row * numBColumns + col] = sum;
    }
}

/**
    A * B = C
    N = numero di colonne
*/

hipError_t matrixMultiplication(float *A, float *B, float *C, int numARows,int numAColumns, int numBRows, int numBColumns){
    hipError_t cudaStatus;
    //@@ Initialize the grid and block dimensions here
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((float)numAColumns) / blockDim.x),ceil(((float)numBRows) / blockDim.y));
    
    /*
    dim3 blockDim(numARows, numBColumns);
    dim3 gridDim(1, 1);
    //<<<blocksPerGrid,threadsPerBlock>>>
    if (numARows * numBColumns > 512){
        blockDim.x = 512;
        blockDim.y = 512;
        gridDim.x = ceil(double(numBColumns)/double(blockDim.x));
        gridDim.y = ceil(double(numARows)/double(blockDim.y));
    }
    */ 
    
    float *d_A, *d_B, *d_C;

    cudaStatus = hipMalloc((void **) &d_A, sizeof(float)*numARows*numAColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void **) &d_B, sizeof(float)*numBRows*numBColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void **) &d_C, sizeof(float)*numARows * numBColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    //copio i vettori
    hipMemcpy(d_A,A,sizeof(float)*numARows*numAColumns,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,sizeof(float)*numBRows*numBColumns,hipMemcpyHostToDevice);
    
   

    hipMemset(d_C, 0, numARows * numBColumns * sizeof(float));

    generic_mat_mul<<<gridDim, blockDim>>>(d_A, d_B, d_C, numARows, numAColumns, numBRows, numBColumns);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(C, d_C,numARows * numBColumns * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    //@@ Free the GPU memory here
Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return cudaStatus;
    
}