#include "hip/hip_runtime.h"
#include "../lib/utilities.h"

using namespace std;
using namespace cv;

void stampaMatrice(float *matrice, int rig, int col){
    //stampa a matrice
    for(int i = 0; i < rig; i++){
        for(int j = 0; j < col; j++){
            
            cout << matrice[i * col + j] << "\t";
        }
        cout<<"\n";
    }
    cout<<"\n\n";
    
}

/**

*/
__global__ void generic_mat_mul(float *A, float *B, float *C, int numARows,int numAColumns, int numBRows, int numBColumns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numARows && col < numBColumns) {
        float sum = 0;
        for (int ii = 0; ii < numAColumns; ii++) {
            sum += A[row * numAColumns + ii] * B[ii * numBColumns + col];
        }
        C[row * numBColumns + col] = sum;
    }
}

/**
    A * B = C
    N = numero di colonne
*/

hipError_t matrixMultiplication(float *A, float *B, float *C, int numARows,int numAColumns, int numBRows, int numBColumns){
    hipError_t cudaStatus;
    //@@ Initialize the grid and block dimensions here
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((float)numAColumns) / blockDim.x),ceil(((float)numBRows) / blockDim.y));
    
    /*
    dim3 blockDim(numARows, numBColumns);
    dim3 gridDim(1, 1);
    //<<<blocksPerGrid,threadsPerBlock>>>
    if (numARows * numBColumns > 512){
        blockDim.x = 512;
        blockDim.y = 512;
        gridDim.x = ceil(double(numBColumns)/double(blockDim.x));
        gridDim.y = ceil(double(numARows)/double(blockDim.y));
    }
    */ 
    
    float *d_A, *d_B, *d_C;

    cudaStatus = hipMalloc((void **) &d_A, sizeof(float)*numARows*numAColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void **) &d_B, sizeof(float)*numBRows*numBColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void **) &d_C, sizeof(float)*numARows * numBColumns);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    //copio i vettori
    hipMemcpy(d_A,A,sizeof(float)*numARows*numAColumns,hipMemcpyHostToDevice);
    
    hipMemcpy(d_B,B,sizeof(float)*numBRows*numBColumns,hipMemcpyHostToDevice);
    
   

    hipMemset(d_C, 0, numARows * numBColumns * sizeof(float));

    generic_mat_mul<<<gridDim, blockDim>>>(d_A, d_B, d_C, numARows, numAColumns, numBRows, numBColumns);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(C, d_C,numARows * numBColumns * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    //@@ Free the GPU memory here
Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return cudaStatus;
    
}



void arrayToMat(cv::Mat &mat, float *array, int numElem){
    memcpy(mat.ptr(),array,numElem * sizeof(float));
}


void matToArray(float *array, cv::Mat &mat, int numElem){
    memcpy(array,mat.ptr(),numElem * sizeof(float));
}


// implementazione manuale di

//---------------------------- cv::warpPerspective() ---------------------------------------

//  warpPerspective(InputArray src, OutputArray dst, InputArray M, Size dsize, int flags=INTER_LINEAR, int borderMode=BORDER_CONSTANT, const Scalar& borderValue=Scalar())

// M è sempre 3 x 3 
// dsize è la dimensione dell'immagine sorgente
// hipError_t warpPerspectiveCPU(float *src, float *dst, float *m, int numSrcRows,int numSrcColumns){
    
//     for(int i = 0; i < rig; i++){
//         for(int j = 0; j < col; j++){
            
//             //cout << matrice[i * col + j] << "\t";
//             dst[i * col *j] = src[i * col *j]
//         }
//         cout<<"\n";
//     }
// }


// Mat A = immagine da traslare
// Mat H = matrice di transformazione (3 X 3)

Mat warpPerspectiveCPU(Mat A, Mat H){
    // allocate array of all locations
    int Numrows = A.rows;
    int Numcols = A.cols;
    int channels   = A.channels();
    // cout << "rows " << Numrows << "col " << Numcols << "channels " << channels <<endl;
    int size = Numrows*Numcols;
    int MaxX,MaxY = -1000;
    int MinX,MinY =  1000;
    int *TransArry = (int *)malloc(sizeof(int)*size);
    int Idx;

    int homeX=Idx % Numcols;
    int homeY=Idx / Numcols;
    // cout << H << endl;

    //waitKey();         
    for (Idx=0; Idx < size; ++Idx ){

        homeX=Idx % Numcols;
        homeY=Idx / Numcols;

        float x  = (H.at<float>(0,0) * (homeX)) +( H.at<float>(0,1) * (homeY)) + ( H.at<float>(0,2) * 1) ;
        float y  = (H.at<float>(1,0) * (homeX)) +( H.at<float>(1,1) * (homeY)) + ( H.at<float>(1,2) * 1) ;
        float s  = (H.at<float>(2,0) * (homeX)) +( H.at<float>(2,1) * (homeY)) + ( H.at<float>(2,2) * 1) ;

        // cout << " x = " << x << " y= " << y << " s= " << s;
        x = floor(x/s);

        y = floor(y/s);

        // for the first col in TransMatrix
        if (homeX ==0){
            if (x > MaxX) MaxX = x;
            if (x < MinX) MinX = x; 
        }

        //for thee first row in TransMatrix
        if (homeY ==0){
            if (y > MaxY) MaxY = y;
            if (y < MinY) MinY = y;
        }
        if((y)>=A.rows || (y)<0 || (x)>=A.cols || (x)<0){
            TransArry[Idx]  = -1;
            // cout << "x= " << x << "y= "<< y << endl;
        }else{
            TransArry[Idx] = (y * Numcols + x); 
        }           

        //cout << Numcols << endl;
        // cout <<     "New index of " << Idx << "is " << TransArry[Idx] << endl;
    }

    Mat   tranImg ;

    A.copyTo(tranImg);
    tranImg = tranImg - tranImg;
    // cout <<     "Rows" << tranImg.rows << "cols" << tranImg.cols << "cha" <<  A.channels() << endl;


    //waitKey();
    // Remap Image
    for (Idx=0; Idx < size; Idx ++ ){

        homeX=Idx % Numcols;
        homeY=Idx / Numcols;                
        //tranImg.at<uchar>(homeY, homeX) =0;
        if(TransArry[Idx] != -1){   
            //cout << "Index " << Idx << "Passed " << endl;
            int newhomeX=TransArry[Idx] % Numcols; // Col ID
            int newhomeY=TransArry[Idx] / Numcols;  // Row ID


            // cout << "Index is " << Idx << endl;
            // cout << "HomeX is " << homeX << " and HomeY is " << homeY << endl;
            // cout << "New Index is " << TransArry[Idx] << endl;
            // cout << "New HomeX is " << newhomeX << " and New HomeY is " << newhomeY << endl;   
            // cout << "*****************************************"<< endl; 
            // if (!(Idx%100)) sleep(20);  

            tranImg.at<uchar>(newhomeY, (newhomeX*channels)) = A.at<uchar>(homeY, homeX*channels);
            if(channels>1)
                tranImg.at<uchar>(newhomeY, newhomeX*channels+1) = A.at<uchar>(homeY, homeX*channels+1);
            if(channels>2)
                tranImg.at<uchar>(newhomeY, newhomeX*channels+2) = A.at<uchar>(homeY, homeX*channels+2);
            // if (!(Idx%100)){
                // imshow("inside", tranImg);
                // waitKey(1);
                // }
            }
    }
    //cout << tranImg << endl;  
    
    return tranImg;

}

