#include "hip/hip_runtime.h"
#include "../lib/utilities.h"
using namespace std;
using namespace cv;

//typedef unsigned char uchar;

vector<Mat> imageSplitting(Mat image){
     
    Mat Bands[3],merged;
    split(image, Bands);
    vector<Mat> channels = {Bands[0],Bands[1],Bands[2]};
    // merge(channels,merged);
    // imshow("red", Bands[2]);
    // imshow("blue", Bands[0]);
    // imshow("green",Bands[1]);
    // imshow("merged",merged);

    return channels;
}


void cvMatToVector(Mat mat, uchar *v){
    cout << "cvMatToVector image :\n \t \ttipo matrice :" << "CV_" + type2str(mat.type()) <<endl;

    std::vector<uchar> array;
    cout << "mat size : " << mat.rows * mat.cols <<"\n";
    
    if (mat.isContinuous()) {
        array.assign(mat.data, mat.data + mat.total());
    }else{
        for (int i = 0; i < mat.rows; ++i) {
            array.insert(array.end(), mat.ptr<uchar>(i), mat.ptr<uchar>(i)+mat.cols);
        }
    }
    /*
    for(int r = 0; r < mat.rows; r++){
        for(int c = 0; c < mat.rows; c++){
            array.push_back(mat.at<cv::Vec3b>(r,c)[0]);
            array.push_back(mat.at<cv::Vec3b>(r,c)[1]);
            array.push_back(mat.at<cv::Vec3b>(r,c)[2]); 
        }
    }
    */

    v = (uchar *)malloc(sizeof(uchar)*array.size());
    cout << "array size : " << array.size() <<endl;
    //getchar();
    for (int i = 0; i <array.size(); i++){
        v[i] = array[i];
    }

}

std::vector<uchar> multiChannelMatToVector(Mat mat){
    std::vector<uchar> array;
    cout << "size : "<< mat.size() << " num canali : "<< mat.channels() << endl;
    /*
    for(int rows = 0; rows < mat.rows; rows++){
        for(int cols = 0; cols < mat.cols * mat.channels(); cols++){
            array.push_back(mat.at<uchar>(rows, cols));

        }
    }
    */
    if (mat.isContinuous()) {
        array.assign(mat.data, mat.data + mat.total());
    }else{
        for (int i = 0; i < mat.rows; ++i) {
            array.insert(array.end(), mat.ptr<uchar>(i), mat.ptr<uchar>(i)+mat.cols);
        }
    }
    
    /*
    cout << "\nmat.at<uchar>(0, 0) = " << (int) mat.at<uchar>(0, 0) 
         << " mat.at<uchar>(0, 1) = " << (int) mat.at<uchar>(0, 1) 
         << " mat.at<uchar>(0, 2) = " << (int) mat.at<uchar>(0, 2) << endl;
    cout << "\n v[0] = " << (int) array[0] 
         << " v[1] = " << (int) array[1]
         << " v[2] = " << (int) array[2] << endl;

    cout << "\nmat.at<uchar>(0, 3) = " << (int) mat.at<uchar>(0, 0) 
         << " mat.at<uchar>(0, 4) = " << (int) mat.at<uchar>(0, 1) 
         << " mat.at<uchar>(0, 5) = " << (int) mat.at<uchar>(0, 2) << endl;
    cout << "\n v[3] = " << (int) array[0] 
         << " v[4] = " << (int) array[1]
         << " v[5] = " << (int) array[2] << endl;
    */
    return array;
}

void stdVectorToArray(std::vector<uchar> &input, uchar *out){
    //out = (uchar *)malloc(sizeof(uchar)*input.size());
    cout << " elementi del vettore : "<< input.size() << endl;
    for(int i = 0; i < input.size(); i++){
        out[i] = input[i];
    }
}



__global__ void remapping_single_ch_image_cuda_kernel(uchar *image, int numRows, int numCols, int *tranfArray, int numChannel, uchar *output){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * numCols + col;
    int homeX, homeY;
    int newhomeX, newhomeY;
    if (idx < numCols * numRows){
    //if (col < numCols && row < numRows){
        homeX=idx % numCols;
        homeY=idx / numCols; 
        if(tranfArray[idx] != -1 ){  
        //if(tranfArray[idx] != -1 && (homeY * numCols + homeX) < (numCols * numRows)){   
            //cout << "Index " << Idx << "Passed " << endl;
            newhomeX = tranfArray[idx] % numCols; // Col ID
            newhomeY = tranfArray[idx] / numCols;  // Row ID

            //i * col + j
            output[newhomeY * numCols + newhomeX] = image [homeY * numCols + homeX];
            
            
        }
    }
    // codice da parallelizzare :

    // Remap Image
    // for (Idx=0; Idx < size; Idx ++ ){

    //     homeX=Idx % Numcols;
    //     homeY=Idx / Numcols;                
    //     //tranImg.at<uchar>(homeY, homeX) =0;
    //     if(TransArry[Idx] != -1){   
    //         //cout << "Index " << Idx << "Passed " << endl;
    //         int newhomeX=TransArry[Idx] % Numcols; // Col ID
    //         int newhomeY=TransArry[Idx] / Numcols;  // Row ID
    //         tranImg.at<uchar>(newhomeY, (newhomeX*channels)) = input.at<uchar>(homeY, homeX*channels);
    //         if(channels>1)
    //             tranImg.at<uchar>(newhomeY, newhomeX*channels+1) = input.at<uchar>(homeY, homeX*channels+1);
    //         if(channels>2)
    //             tranImg.at<uchar>(newhomeY, newhomeX*channels+2) = input.at<uchar>(homeY, homeX*channels+2);
            
    //         }
    // }

}


__global__ void remapping_multi_ch_image_cuda_kernel(uchar *image, int numRows, int numCols, int *tranfArray, int numChannel, uchar *output){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int idx = row * numCols + col;
    int homeX, homeY;
    int newhomeX, newhomeY;
    //if (idx < numCols * numRows){
    if (idx < numCols * numRows){
    //if (col < numCols*numChannel && row < numRows){
        homeX=idx % numCols;
        homeY=idx / numCols; 
        if(tranfArray[idx] != -1 ){   
            //cout << "Index " << Idx << "Passed " << endl;
            newhomeX = tranfArray[idx] % numCols; // Col ID
            newhomeY = tranfArray[idx] / numCols;  // Row ID

            //i * col + j
            output[newhomeY * numCols + (newhomeX * numChannel)] = image [homeY * numCols + (homeX* numChannel)]; // B
            
            if(numChannel > 1)
                output[(newhomeY * numCols) + (newhomeX * numChannel + 1)] = image [(homeY * numCols) + (homeX * numChannel + 1)]; // G
            if(numChannel > 2)
                output[(newhomeY * numCols) + (newhomeX * numChannel + 2)] = image [(homeY * numCols) + (homeX * numChannel + 2)]; // R
            
        }
    }
    

}



/**
    restituisce in output l'immagine rimappata
*/
cv::Mat remappingSingleChannelImage(Mat image, int *tranfArray){
    hipError_t cudaStatus;
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((float)image.cols) / blockDim.x),ceil(((float)image.rows) / blockDim.y));

    int size = image.rows * image.cols;
    Mat img = image.clone();
    // Mat img = Mat::zeros(cv::Size(image.rows, image.cols), CV_32FC3);
    // img = image.clone();

    
	cout << "Remapping image :\n \t \ttipo matrice :" << "CV_" + type2str(image.type()) <<endl;

    //cout <<"\n (float *)malloc(sizeof(float)*size) ";
    uchar *h_image = (uchar *)malloc(sizeof(uchar)*size);
    
    uchar *d_image, *d_output;

    int *d_tranfArray;
    cout <<" \n alloco il vettore sul device per l'immagine \n";
    cudaStatus = hipMalloc((void **) &d_image, sizeof(uchar) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorRemapping;
    }
    cout <<" \n alloco il vettore immagine per l'output\n";
    cudaStatus = hipMalloc((void **) &d_output, sizeof(uchar) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorRemapping;
    }
    cout <<" \n alloco il vettore di transposizione \n";
    cudaStatus = hipMalloc((void **) &d_tranfArray, sizeof(int) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorRemapping;
    }

    //matToArray(h_image, image, image.rows, image.cols);
    //std::memcpy( h_image,img.data, size*sizeof(uchar));
    cvMatToVector(img, h_image);


    //copio i vettori
    cudaStatus = hipMemcpy(d_image,h_image,sizeof(uchar) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemCpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorRemapping;
    }

    cudaStatus = hipMemcpy(d_tranfArray,tranfArray,sizeof(int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemSetfailed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorRemapping;
    }



    //__global__ void remapping_image_cuda_kernel(float *image, int numRows, int numCols, int *tranfArray, int numChannel, float *output){
    cout<<"\n RICHIAMO IL KERNELL PER IL REMAPPING DELL'IMMAGINE \n";
    //   <<<gridDim, blockDim>>>
    //remapping_single_ch_image_cuda_kernel<<<ceil(size/256.0),256>>>(d_image, image.rows, image.cols, d_tranfArray, image.channels(),d_output);
    remapping_single_ch_image_cuda_kernel<<<gridDim, blockDim>>>(d_image, image.rows, image.cols, d_tranfArray, image.channels(),d_output);
    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorRemapping;
    }

    cout <<" \n copio il risultato del kernel \n";
    cudaStatus = hipMemcpy(h_image,d_output,sizeof(uchar) * size,hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemCpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorRemapping;
    }
    /**
    * converte un vettore in un oggetto Mat
    * src : array
    * dst : Mat
    */
    cout <<" \n copio il risultato del kernel nell'oggetto mat\n";
    //arrayToMat(img, h_image, size);
    //memcpy(img.data())
    std::memcpy(img.data, h_image, size*sizeof(uchar));
    cout <<" \n finita la copia \n";

    return img;



ErrorRemapping:
    //cout<< "****** ERRORE CUDA ****** : " << cudaStatus << endl;
    hipFree(d_image);
    hipFree(d_output);
    hipFree(d_tranfArray);
    return Mat::zeros(cv::Size(image.rows, image.cols), CV_8UC1);
    

}


cv::Mat OLDremappingMultiChannelImage(Mat image, int *tranfArray){
    hipError_t cudaStatus;
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((float)image.cols) / blockDim.x),ceil(((float)image.rows) / blockDim.y));
    int num_RGBelem,size = image.rows * image.cols;
    Mat img = image.clone();
    // Mat img = Mat::zeros(cv::Size(image.rows, image.cols), CV_32FC3);
    // img = image.clone();
	cout << "Remapping image :\n \t \ttipo matrice :" << "CV_" + type2str(image.type()) <<endl;
     
    uchar *d_image, *d_output;
    int *d_tranfArray;
    vector<uchar> image_array;

    image_array = multiChannelMatToVector(img);
    //conto il numero di elementi totali
    num_RGBelem = image_array.size();

    uchar *h_image = (uchar *)malloc(sizeof(uchar)*num_RGBelem);
    
    cout << "num_RGBelem : "<< num_RGBelem << endl;

    

    // alloco la memoria sulla GPU
    //cout <<" \n alloco il vettore sul device per l'immagine \n";
    cudaStatus = hipMalloc((void **) &d_image, sizeof(uchar) * num_RGBelem);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorMultiRemapping;
    }
    //cout <<" \n alloco il vettore immagine per l'output\n";
    cudaStatus = hipMalloc((void **) &d_output, sizeof(uchar) * num_RGBelem);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorMultiRemapping;
    }
    //cout <<" \n alloco il vettore di transposizione \n";
    cudaStatus = hipMalloc((void **) &d_tranfArray, sizeof(int) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorMultiRemapping;
    }

    //traduco lo std::vector in un array normale
    stdVectorToArray(image_array, h_image);

    //copio i dati sulla GPU

    //copio i vettori
    cudaStatus = hipMemcpy(d_image,h_image,sizeof(uchar) * num_RGBelem, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemCpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorMultiRemapping;
    }

    cudaStatus = hipMemcpy(d_tranfArray,tranfArray,sizeof(int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemSetfailed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorMultiRemapping;
    }

    // richiamo il cuda kernel
    remapping_multi_ch_image_cuda_kernel<<<gridDim,blockDim>>>(d_image, image.rows, image.cols , d_tranfArray, image.channels(),d_output);
    //remapping_multi_ch_image_cuda_kernel<<<ceil((float)(num_RGBelem/256.0)),256>>>(d_image, image.rows, image.cols , d_tranfArray, image.channels(),d_output);
    //remapping_multi_ch_image_cuda_kernel<<<ceil(num_RGBelem/256.0),256>>>(d_image, image.rows, image.cols * image.channels(), d_tranfArray, image.channels(),d_output);
    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorMultiRemapping;
    }

    cout <<" \n copio il risultato del kernel \n";
    cudaStatus = hipMemcpy(h_image,d_output,sizeof(uchar) * num_RGBelem,hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemCpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorMultiRemapping;
    }
    /**
    * converte un vettore in un oggetto Mat
    * src : array
    * dst : Mat
    */
    cout <<" \n copio il risultato del kernel nell'oggetto mat\n";
    //arrayToMat(img, h_image, size);
    //memcpy(img.data())
    memcpy(img.data, h_image, num_RGBelem*sizeof(uchar));
    cout <<" \n finita la copia \n";

    return img;



ErrorMultiRemapping:
    //cout<< "****** ERRORE CUDA ****** : " << cudaStatus << endl;
    hipFree(d_image);
    hipFree(d_output);
    hipFree(d_tranfArray);
    //return Mat::zeros(cv::Size(image.rows, image.cols), CV_8UC1);
    exit(0);

}
// __global__ void new_remapping_kernel(uchar* src, int numRows, int numCols, size_t step, int numChannel, int *tranfArray, uchar* out){

__global__ void new_remapping_kernel(cv::cuda::PtrStepSz<uchar3> src, int numRows, int numCols, size_t step, int numChannel, int *tranfArray, cv::cuda::PtrStepSz<uchar3> out){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int idx = row * numCols + col;
    int homeX, homeY;
    int newhomeX, newhomeY;
    int iStep = step / sizeof(uchar3);
    int oStep = step / sizeof(uchar3);
    uchar3 pxval;


    if ((row < numRows) && (col < numCols))
    {
        // azzero il pixell attuale
        // uchar * px = out + (row * step);
        // px[col] = 0; px[col+1] = 0; px[col+1] = 0;

        homeX=idx % numCols;
        homeY=idx / numCols; 
        if(tranfArray[idx] != -1 ){ 
            newhomeX = tranfArray[idx] % numCols; // Col ID
            newhomeY = tranfArray[idx] / numCols;  // Row ID
            //uchar *outrowptr = out + newhomeY * step;
            //uchar *srcrowptr = src + homeY * step;
            
            // outrowptr[newhomeX] = srcrowptr[homeX];
            // out(newhomeY, newhomeX*numChannel) = src(homeY, homeX*numChannel);
            // out(newhomeY, newhomeX) = src(homeY, homeX);
            pxval = src(homeY, homeX );
            out(newhomeY, newhomeX) = pxval;
            
            if (numChannel > 1){
                pxval = src(homeY, homeX  + 1);
                out(newhomeY, newhomeX  + 1) = pxval;
            }
            
            if (numChannel > 2){
                // outrowptr[newhomeX * numChannel + 2] = srcrowptr[homeX * numChannel + 2];
                // outrowptr[newhomeX + 2] = srcrowptr[homeX + 2];
                // out(newhomeY, newhomeX*numChannel + 2) = src(homeY, homeX*numChannel+2);
                // out(newhomeY, newhomeX + 2) = src(homeY, homeX + 2);
                pxval = src(homeY, homeX  +2);
                out(newhomeY, newhomeX  + 2) = pxval;
            }
        }
    

    }
}

cv::Mat remappingMultiChannelImage(Mat image, int *tranfArray){
    hipError_t cudaStatus;
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil((float)image.cols / blockDim.x), ceil((float)image.cols / blockDim.y), 1);
    //dim3 gridDim(ceil(((float)image.cols) / blockDim.x),ceil(((float)image.rows) / blockDim.y));
    int num_RGBelem,size = image.rows * image.cols;
    // Mat img = image.clone();
    Mat null_mat = Mat::zeros(cv::Size(image.rows, image.cols), CV_8UC3);
    // img = image.clone();
    cout << "Remapping image :\n \t \ttipo matrice :" << "CV_" + type2str(image.type()) <<endl;
    cv::Mat img;
    uchar *d_image, *d_output;
    int *d_tranfArray;
    // vector<uchar> image_array;

    //definisco l'immagine 
    cv::cuda::GpuMat input, output;
    input.upload(image);
    //output.create(cv::Size(image.rows, image.cols), CV_8UC3);
    //output = input.clone();
    output.upload(null_mat);
    //cout << "Remapping image :\n \t \ttipo matrice :" << "CV_" + type2str(input.type()) <<endl;

    //cout <<" \n alloco il vettore di transposizione \n";
    cudaStatus = hipMalloc((void **) &d_tranfArray, sizeof(int) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorNewMultiRemapping;
    }

    //cout <<" \n copio il vettore di transposizione \n";
    cudaStatus = hipMemcpy(d_tranfArray,tranfArray,sizeof(int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemSetfailed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorNewMultiRemapping;
    }
    
    cout << "\n step / sizeof(uchar3) = " << ( int )input.step / sizeof(uchar3) << endl;

    new_remapping_kernel<<<gridDim,blockDim>>> (input, input.rows, input.cols, input.step, image.channels(), d_tranfArray, output);
    hipDeviceSynchronize();
    
    output.download(img);

    return img;

ErrorNewMultiRemapping:
    
    hipFree(d_tranfArray);
    //return Mat::zeros(cv::Size(image.rows, image.cols), CV_8UC1);
    exit(0);

}




hipError_t warpPerspectiveRemappingCUDA(Mat input, Mat &output, const Mat H){
    hipError_t cudaStatus;

    // allocate array of all locations
    int Numrows = input.rows;
    int Numcols = input.cols;
    int channels   = input.channels();
    // cout << "rows " << Numrows << "col " << Numcols << "channels " << channels <<endl;
    int size = Numrows*Numcols;
    // int MaxX,MaxY = -1000;
    // int MinX,MinY =  1000;
    
    // int Idx;
    // int homeX, homeY;
    int *TransArry = (int *)malloc(sizeof(int)*size);
    // float *d_H;
    // float *vecH = (float *)malloc(sizeof(float) * H.rows * H.cols);
    // int *d_T;

    Mat tranImg;
    
    cudaStatus = calculateTransferArray(H,TransArry,Numrows, Numcols);


    //input.copyTo(tranImg);
    input.copyTo(tranImg);
    tranImg = tranImg - tranImg;
    
    //cv::Mat remappingImage(Mat &image, int *tranfArray)
    
    cout <<" \n richiamo la funzione per il remapping \n";
    cout <<" \n NUMERO DI CANALI : " << input.channels() << "\n";
    

    //single channel img
    /*
    vector<Mat> splitImg = imageSplitting(input);
    // getchar();
    // Mat merged;
    // imshow("red", splitImg[2]);
    // imshow("blue", splitImg[0]);
    // imshow("green",splitImg[1]);
    // merge(splitImg,merged);
    // imshow("green",merged);
    vector<Mat> result;
    
    // Mat output_blue = remappingImage(splitImg[0], TransArry);
    // Mat output_green = remappingImage(splitImg[1], TransArry);
    // Mat output_red = remappingImage(splitImg[2], TransArry);
    result.push_back(remappingSingleChannelImage(splitImg[0], TransArry));
    result.push_back(remappingSingleChannelImage(splitImg[1], TransArry));
    result.push_back(remappingSingleChannelImage(splitImg[2], TransArry));

    merge(result,output);
    */
    
    // output = OLDremappingMultiChannelImage(input, TransArry);
    output = remappingMultiChannelImage(input, TransArry);

    return cudaStatus;
}





