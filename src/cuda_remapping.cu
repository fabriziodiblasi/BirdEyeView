#include "hip/hip_runtime.h"
#include "../lib/utilities.h"
using namespace std;
using namespace cv;

typedef unsigned char uchar;

vector<Mat> imageSplitting(Mat image){
     
    Mat Bands[3],merged;
    split(image, Bands);
    vector<Mat> channels = {Bands[0],Bands[1],Bands[2]};
    // merge(channels,merged);
    // imshow("red", Bands[2]);
    // imshow("blue", Bands[0]);
    // imshow("green",Bands[1]);
    // imshow("merged",merged);

    return channels;
}


void cvMatToVector(Mat mat, uchar *v){
    cout << "cvMatToVector image :\n \t \ttipo matrice :" << "CV_" + type2str(mat.type()) <<endl;

    std::vector<uchar> array;
    cout << "mat size : " << mat.rows * mat.cols <<"\n";
    
    if (mat.isContinuous()) {
        array.assign(mat.data, mat.data + mat.total());
    }else{
        for (int i = 0; i < mat.rows; ++i) {
            array.insert(array.end(), mat.ptr<uchar>(i), mat.ptr<uchar>(i)+mat.cols);
        }
    }
    /*
    for(int r = 0; r < mat.rows; r++){
        for(int c = 0; c < mat.rows; c++){
            array.push_back(mat.at<cv::Vec3b>(r,c)[0]);
            array.push_back(mat.at<cv::Vec3b>(r,c)[1]);
            array.push_back(mat.at<cv::Vec3b>(r,c)[2]); 
        }
    }
    */

    v = (uchar *)malloc(sizeof(uchar)*array.size());
    cout << "array size : " << array.size() <<endl;
    //getchar();
    for (int i = 0; i <array.size(); i++){
        v[i] = array[i];
    }

}

std::vector<uchar> multiChannelMatToVector(Mat mat){
    std::vector<uchar> array;
    cout << "size : "<< mat.size() << " num canali : "<< mat.channels() << endl;
    
    for(int rows = 0; rows < mat.rows; rows++){
        for(int cols = 0; cols < mat.cols * mat.channels(); cols++){
            array.push_back(mat.at<uchar>(rows, cols));

        }
    }
    /*
    if (mat.isContinuous()) {
        array.assign(mat.data, mat.data + mat.total());
    }else{
        for (int i = 0; i < mat.rows; ++i) {
            array.insert(array.end(), mat.ptr<uchar>(i), mat.ptr<uchar>(i)+mat.cols);
        }
    }
    */
    /*
    cout << "\nmat.at<uchar>(0, 0) = " << (int) mat.at<uchar>(0, 0) 
         << " mat.at<uchar>(0, 1) = " << (int) mat.at<uchar>(0, 1) 
         << " mat.at<uchar>(0, 2) = " << (int) mat.at<uchar>(0, 2) << endl;
    cout << "\n v[0] = " << (int) array[0] 
         << " v[1] = " << (int) array[1]
         << " v[2] = " << (int) array[2] << endl;

    cout << "\nmat.at<uchar>(0, 3) = " << (int) mat.at<uchar>(0, 0) 
         << " mat.at<uchar>(0, 4) = " << (int) mat.at<uchar>(0, 1) 
         << " mat.at<uchar>(0, 5) = " << (int) mat.at<uchar>(0, 2) << endl;
    cout << "\n v[3] = " << (int) array[0] 
         << " v[4] = " << (int) array[1]
         << " v[5] = " << (int) array[2] << endl;
    */
    return array;
}

void stdVectorToArray(std::vector<uchar> &input, uchar *out){
    //out = (uchar *)malloc(sizeof(uchar)*input.size());
    cout << " elementi del vettore : "<< input.size() << endl;
    for(int i = 0; i < input.size(); i++){
        out[i] = input[i];
    }
}



__global__ void remapping_single_ch_image_cuda_kernel(uchar *image, int numRows, int numCols, int *tranfArray, int numChannel, uchar *output){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * numCols + col;
    int homeX, homeY;
    int newhomeX, newhomeY;
    if (idx < numCols * numRows){
    //if (col < numCols && row < numRows){
        homeX=idx % numCols;
        homeY=idx / numCols; 
        if(tranfArray[idx] != -1 ){  
        //if(tranfArray[idx] != -1 && (homeY * numCols + homeX) < (numCols * numRows)){   
            //cout << "Index " << Idx << "Passed " << endl;
            newhomeX = tranfArray[idx] % numCols; // Col ID
            newhomeY = tranfArray[idx] / numCols;  // Row ID

            //i * col + j
            output[newhomeY * numCols + newhomeX] = image [homeY * numCols + homeX];
            
            
        }
    }
    // codice da parallelizzare :

    // Remap Image
    // for (Idx=0; Idx < size; Idx ++ ){

    //     homeX=Idx % Numcols;
    //     homeY=Idx / Numcols;                
    //     //tranImg.at<uchar>(homeY, homeX) =0;
    //     if(TransArry[Idx] != -1){   
    //         //cout << "Index " << Idx << "Passed " << endl;
    //         int newhomeX=TransArry[Idx] % Numcols; // Col ID
    //         int newhomeY=TransArry[Idx] / Numcols;  // Row ID
    //         tranImg.at<uchar>(newhomeY, (newhomeX*channels)) = input.at<uchar>(homeY, homeX*channels);
    //         if(channels>1)
    //             tranImg.at<uchar>(newhomeY, newhomeX*channels+1) = input.at<uchar>(homeY, homeX*channels+1);
    //         if(channels>2)
    //             tranImg.at<uchar>(newhomeY, newhomeX*channels+2) = input.at<uchar>(homeY, homeX*channels+2);
            
    //         }
    // }

}


__global__ void remapping_multi_ch_image_cuda_kernel(uchar *image, int numRows, int numCols, int *tranfArray, int numChannel, uchar *output){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int idx = row * numCols + col;
    int homeX, homeY;
    int newhomeX, newhomeY;
    //if (idx < numCols * numRows){
    if (idx < numCols * numRows){
    //if (col < numCols*numChannel && row < numRows){
        homeX=idx % numCols;
        homeY=idx / numCols; 
        if(tranfArray[idx] != -1 ){   
            //cout << "Index " << Idx << "Passed " << endl;
            newhomeX = tranfArray[idx] % numCols; // Col ID
            newhomeY = tranfArray[idx] / numCols;  // Row ID

            //i * col + j
            output[newhomeY * numCols + newhomeX] = image [homeY * numCols + homeX]; // B
            
            if(numChannel > 1)
                output[(newhomeY * numCols) + (newhomeX * numChannel + 1)] = image [(homeY * numCols) + (homeX * numChannel + 1)]; // G
            if(numChannel > 2)
                output[(newhomeY * numCols) + (newhomeX * numChannel + 2)] = image [(homeY * numCols) + (homeX * numChannel + 2)]; // R
            
        }
    }
    

}



/**
    restituisce in output l'immagine rimappata
*/
cv::Mat remappingSingleChannelImage(Mat image, int *tranfArray){
    hipError_t cudaStatus;
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((float)image.cols) / blockDim.x),ceil(((float)image.rows) / blockDim.y));

    int size = image.rows * image.cols;
    Mat img = image.clone();
    // Mat img = Mat::zeros(cv::Size(image.rows, image.cols), CV_32FC3);
    // img = image.clone();

    
	cout << "Remapping image :\n \t \ttipo matrice :" << "CV_" + type2str(image.type()) <<endl;

    //cout <<"\n (float *)malloc(sizeof(float)*size) ";
    uchar *h_image = (uchar *)malloc(sizeof(uchar)*size);
    
    uchar *d_image, *d_output;

    int *d_tranfArray;
    cout <<" \n alloco il vettore sul device per l'immagine \n";
    cudaStatus = hipMalloc((void **) &d_image, sizeof(uchar) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorRemapping;
    }
    cout <<" \n alloco il vettore immagine per l'output\n";
    cudaStatus = hipMalloc((void **) &d_output, sizeof(uchar) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorRemapping;
    }
    cout <<" \n alloco il vettore di transposizione \n";
    cudaStatus = hipMalloc((void **) &d_tranfArray, sizeof(int) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorRemapping;
    }

    //matToArray(h_image, image, image.rows, image.cols);
    //std::memcpy( h_image,img.data, size*sizeof(uchar));
    cvMatToVector(img, h_image);


    //copio i vettori
    cudaStatus = hipMemcpy(d_image,h_image,sizeof(uchar) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemCpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorRemapping;
    }

    cudaStatus = hipMemcpy(d_tranfArray,tranfArray,sizeof(int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemSetfailed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorRemapping;
    }



    //__global__ void remapping_image_cuda_kernel(float *image, int numRows, int numCols, int *tranfArray, int numChannel, float *output){
    cout<<"\n RICHIAMO IL KERNELL PER IL REMAPPING DELL'IMMAGINE \n";
    //   <<<gridDim, blockDim>>>
    //remapping_single_ch_image_cuda_kernel<<<ceil(size/256.0),256>>>(d_image, image.rows, image.cols, d_tranfArray, image.channels(),d_output);
    remapping_single_ch_image_cuda_kernel<<<gridDim, blockDim>>>(d_image, image.rows, image.cols, d_tranfArray, image.channels(),d_output);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorRemapping;
    }

    cout <<" \n copio il risultato del kernel \n";
    cudaStatus = hipMemcpy(h_image,d_output,sizeof(uchar) * size,hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemCpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorRemapping;
    }
    /**
    * converte un vettore in un oggetto Mat
    * src : array
    * dst : Mat
    */
    cout <<" \n copio il risultato del kernel nell'oggetto mat\n";
    //arrayToMat(img, h_image, size);
    //memcpy(img.data())
    std::memcpy(img.data, h_image, size*sizeof(uchar));
    cout <<" \n finita la copia \n";

    return img;



ErrorRemapping:
    //cout<< "****** ERRORE CUDA ****** : " << cudaStatus << endl;
    hipFree(d_image);
    hipFree(d_output);
    hipFree(d_tranfArray);
    return Mat::zeros(cv::Size(image.rows, image.cols), CV_8UC1);
    

}



cv::Mat remappingMultiChannelImage(Mat image, int *tranfArray){
    hipError_t cudaStatus;
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((float)image.cols) / blockDim.x),ceil(((float)image.rows) / blockDim.y));
    int num_RGBelem,size = image.rows * image.cols;
    Mat img = image.clone();
    // Mat img = Mat::zeros(cv::Size(image.rows, image.cols), CV_32FC3);
    // img = image.clone();
	cout << "Remapping image :\n \t \ttipo matrice :" << "CV_" + type2str(image.type()) <<endl;
     
    uchar *d_image, *d_output;
    int *d_tranfArray;
    vector<uchar> image_array;

    image_array = multiChannelMatToVector(img);
    //conto il numero di elementi totali
    num_RGBelem = image_array.size();

    uchar *h_image = (uchar *)malloc(sizeof(uchar)*num_RGBelem);
    
    cout << "num_RGBelem : "<< num_RGBelem << endl;

    

    // alloco la memoria sulla GPU
    //cout <<" \n alloco il vettore sul device per l'immagine \n";
    cudaStatus = hipMalloc((void **) &d_image, sizeof(uchar) * num_RGBelem);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorMultiRemapping;
    }
    //cout <<" \n alloco il vettore immagine per l'output\n";
    cudaStatus = hipMalloc((void **) &d_output, sizeof(uchar) * num_RGBelem);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorMultiRemapping;
    }
    //cout <<" \n alloco il vettore di transposizione \n";
    cudaStatus = hipMalloc((void **) &d_tranfArray, sizeof(int) * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto ErrorMultiRemapping;
    }

    //traduco lo std::vector in un array normale
    stdVectorToArray(image_array, h_image);

    //copio i dati sulla GPU

    //copio i vettori
    cudaStatus = hipMemcpy(d_image,h_image,sizeof(uchar) * num_RGBelem, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemCpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorMultiRemapping;
    }

    cudaStatus = hipMemcpy(d_tranfArray,tranfArray,sizeof(int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemSetfailed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorMultiRemapping;
    }

    // richiamo il cuda kernel
    remapping_multi_ch_image_cuda_kernel<<<gridDim,blockDim>>>(d_image, image.rows, image.cols , d_tranfArray, image.channels(),d_output);
    //remapping_multi_ch_image_cuda_kernel<<<ceil((float)(num_RGBelem/256.0)),256>>>(d_image, image.rows, image.cols , d_tranfArray, image.channels(),d_output);
    //remapping_multi_ch_image_cuda_kernel<<<ceil(num_RGBelem/256.0),256>>>(d_image, image.rows, image.cols * image.channels(), d_tranfArray, image.channels(),d_output);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorMultiRemapping;
    }

    cout <<" \n copio il risultato del kernel \n";
    cudaStatus = hipMemcpy(h_image,d_output,sizeof(uchar) * num_RGBelem,hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaMemCpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto ErrorMultiRemapping;
    }
    /**
    * converte un vettore in un oggetto Mat
    * src : array
    * dst : Mat
    */
    cout <<" \n copio il risultato del kernel nell'oggetto mat\n";
    //arrayToMat(img, h_image, size);
    //memcpy(img.data())
    memcpy(img.data, h_image, num_RGBelem*sizeof(uchar));
    cout <<" \n finita la copia \n";

    return img;



ErrorMultiRemapping:
    //cout<< "****** ERRORE CUDA ****** : " << cudaStatus << endl;
    hipFree(d_image);
    hipFree(d_output);
    hipFree(d_tranfArray);
    //return Mat::zeros(cv::Size(image.rows, image.cols), CV_8UC1);
    exit(0);

}




hipError_t warpPerspectiveRemappingCUDA(Mat input, Mat &output, const Mat H){
    hipError_t cudaStatus;

    // allocate array of all locations
    int Numrows = input.rows;
    int Numcols = input.cols;
    int channels   = input.channels();
    // cout << "rows " << Numrows << "col " << Numcols << "channels " << channels <<endl;
    int size = Numrows*Numcols;
    // int MaxX,MaxY = -1000;
    // int MinX,MinY =  1000;
    
    // int Idx;
    // int homeX, homeY;
    int *TransArry = (int *)malloc(sizeof(int)*size);
    // float *d_H;
    // float *vecH = (float *)malloc(sizeof(float) * H.rows * H.cols);
    // int *d_T;

    Mat tranImg;
    
    cudaStatus = calculateTransferArray(H,TransArry,Numrows, Numcols);


    //input.copyTo(tranImg);
    input.copyTo(tranImg);
    tranImg = tranImg - tranImg;
    
    //cv::Mat remappingImage(Mat &image, int *tranfArray)
    
    cout <<" \n richiamo la funzione per il remapping \n";
    cout <<" \n NUMERO DI CANALI : " << input.channels() << "\n";
    

    //single channel img
    /*
    vector<Mat> splitImg = imageSplitting(input);
    // getchar();
    // Mat merged;
    // imshow("red", splitImg[2]);
    // imshow("blue", splitImg[0]);
    // imshow("green",splitImg[1]);
    // merge(splitImg,merged);
    // imshow("green",merged);
    vector<Mat> result;
    
    // Mat output_blue = remappingImage(splitImg[0], TransArry);
    // Mat output_green = remappingImage(splitImg[1], TransArry);
    // Mat output_red = remappingImage(splitImg[2], TransArry);
    result.push_back(remappingSingleChannelImage(splitImg[0], TransArry));
    result.push_back(remappingSingleChannelImage(splitImg[1], TransArry));
    result.push_back(remappingSingleChannelImage(splitImg[2], TransArry));

    merge(result,output);
    */
    
    output = remappingMultiChannelImage(input, TransArry);
    
    return cudaStatus;
}





