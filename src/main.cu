#include "hip/hip_runtime.h"
#include "../lib/utilities.h"

// namespaces
using namespace std;
using namespace cv;
#define PI 3.1415926


// int frameWidth = 640;
// int frameHeight = 480;
#define FRAMEWIDTH  640
#define FRAMEHEIGHT 480


// ---- GLOBAL VAR ----
int alpha_ = 90, beta_ = 90, gamma_ = 90;
int f_ = 500, dist_ = 500;



string type2str(int type) {
	string r;
  
	uchar depth = type & CV_MAT_DEPTH_MASK;
	uchar chans = 1 + (type >> CV_CN_SHIFT);
  
	switch ( depth ) {
	  case CV_8U:  r = "8U"; break;
	  case CV_8S:  r = "8S"; break;
	  case CV_16U: r = "16U"; break;
	  case CV_16S: r = "16S"; break;
	  case CV_32S: r = "32S"; break;
	  case CV_32F: r = "32F"; break;
	  case CV_64F: r = "64F"; break;
	  default:     r = "User"; break;
	}
  
	r += "C";
	r += (chans+'0');
  
	return r;
}


__global__ void rotation_multiply_kernel(float *d_RX,float *d_RY,float *d_R, int N){
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += d_RX[ROW * N + i] * d_RY[i * N + COL];
        }
    }
    d_R[ROW * N + COL] = tmpSum;

}


void birdsEyeView(const Mat &input, Mat &output){
    double focalLength, dist, alpha, beta, gamma; 

    alpha =((double)alpha_ -90) * PI/180;
    beta =((double)beta_ -90) * PI/180;
    gamma =((double)gamma_ -90) * PI/180;
    focalLength = (double)f_;
    dist = (double)dist_;

    Size input_size = input.size();
    double w = (double)input_size.width, h = (double)input_size.height;


    // Projecion matrix 2D -> 3D
    
    Mat A1 = (Mat_<float>(4, 3)<< 
        1, 0, -w/2,
        0, 1, -h/2,
        0, 0, 0,
        0, 0, 1 );
    
    
    // Rotation matrices Rx, Ry, Rz

    Mat RX = (Mat_<float>(4, 4) << 
        1, 0, 0, 0,
        0, cos(alpha), -sin(alpha), 0,
        0, sin(alpha), cos(alpha), 0,
        0, 0, 0, 1 );

    Mat RY = (Mat_<float>(4, 4) << 
        cos(beta), 0, -sin(beta), 0,
        0, 1, 0, 0,
        sin(beta), 0, cos(beta), 0,
        0, 0, 0, 1	);

    Mat RZ = (Mat_<float>(4, 4) << 
        cos(gamma), -sin(gamma), 0, 0,
        sin(gamma), cos(gamma), 0, 0,
        0, 0, 1, 0,
        0, 0, 0, 1	);

    // R - rotation matrix
    Mat R = RX * RY * RZ;



    // T - translation matrix
    Mat T = (Mat_<float>(4, 4) << 
        1, 0, 0, 0,  
        0, 1, 0, 0,  
        0, 0, 1, dist,  
        0, 0, 0, 1); 
    
    // K - intrinsic matrix 
    Mat K = (Mat_<float>(3, 4) << 
        focalLength, 0, w/2, 0,
        0, focalLength, h/2, 0,
        0, 0, 1, 0
        ); 


    Mat transformationMat = K * (T * (R * A1));

    cout<< " transformationMat : \n "<< transformationMat << endl;

    //cout<< "transformationMat.rows : " << transformationMat.rows << "\ttransformationMat.cols : " << transformationMat.cols << endl;
    //cout << "tipo matrice di transformazione : "<< "CV_" + type2str( transformationMat.type()) << endl;

    warpPerspective(input, output, transformationMat, input_size, INTER_CUBIC | WARP_INVERSE_MAP);

    return;



}





void CUDA_birdsEyeView(const Mat &input, Mat &output){

    hipError_t error;

    double focalLength, dist, alpha, beta, gamma; 

    alpha =((double)alpha_ -90) * PI/180;
    beta =((double)beta_ -90) * PI/180;
    gamma =((double)gamma_ -90) * PI/180;
    focalLength = (double)f_;
    dist = (double)dist_;

    Size input_size = input.size();
    double w = (double)input_size.width, h = (double)input_size.height;
    /*
    compito :
    parallelizzare la funzione birdsEyeView
    aggiungere il file che fa il prodotto tra matrici in cuda
    */

    float A1[12] = {
        1, 0, -w/2,
        0, 1, -h/2,
        0, 0, 0,
        0, 0, 1 
    };



    float RX[16] = {
        1, 0, 0, 0,
        0, cos(alpha), -sin(alpha), 0,
        0, sin(alpha), cos(alpha), 0,
        0, 0, 0, 1 
    };

    float RY[16] ={
        cos(beta), 0, -sin(beta), 0,
        0, 1, 0, 0,
        sin(beta), 0, cos(beta), 0,
        0, 0, 0, 1
    };

    float RZ[16] = {
        cos(gamma), -sin(gamma), 0, 0,
        sin(gamma), cos(gamma), 0, 0,
        0, 0, 1, 0,
        0, 0, 0, 1
    };

    // cout << "stampo RX \n";
    // stampaMatrice(RX , 4, 4);
    // R - rotation matrix
    // Mat R = RX * RY * RZ;

    float R[16], XY[16];
    error = matrixMultiplication(RX, RY, XY, 4, 4, 4, 4);
    if (error != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        exit(0);
    }
    error = matrixMultiplication(XY, RZ, R, 4, 4, 4, 4);
    if (error != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        exit(0);
    }
    /*
    cout << "stampo R \n";
    stampaMatrice(R, 4, 4);
    */
    // T - translation matrix
    float T[16] = { 
        1, 0, 0, 0,  
        0, 1, 0, 0,  
        0, 0, 1, dist,  
        0, 0, 0, 1
    }; 
    // K - intrinsic matrix 
    float K[12] = { 
        focalLength, 0, w/2, 0,
        0, focalLength, h/2, 0,
        0, 0, 1, 0
    };

    //Mat transformationMat = K * (T * (R * A1));
    float R_A1[12], T_RA1[12], transformationvector[9];

    error = matrixMultiplication(R, A1, R_A1, 4, 4, 4, 3);
    if (error != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        exit(0);
    }

    error = matrixMultiplication(T, R_A1, T_RA1, 4, 4, 4, 3);
    if (error != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        exit(0);
    }

    error = matrixMultiplication(K, T_RA1, transformationvector, 4, 4, 4, 4);
    if (error != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        exit(0);
    }

    cv::Mat tranf_mat(3,3,CV_32FC1);

    arrayToMat(tranf_mat,transformationvector,9);
    cout << "matrice di transformazione : \n" << tranf_mat << endl;

    //DA ELIMINARE --- SOLO A SCOPO DI DEBUG
    //output=input.clone();
    warpPerspective(input, output, tranf_mat, input_size, INTER_CUBIC | WARP_INVERSE_MAP);

    return;

}




int main(int argc, char const *argv[]) {
	
	if(argc > 2) {
      cerr << "Usage: " << argv[0] << " ' /path/to/video/ '  or nothing " << endl;
      cout << "Exiting...." << endl;
      return -1;
    }
    int flag=0;
    Mat image,output;
    

    VideoCapture capture;

    if (argc == 1){
        capture.open(0);
    }else{
        string filename = argv[1];
        capture.open(filename);
    }

    if(!capture.isOpened()) throw "Error reading video";

    

    /*
        definisco i parametri e le trackbar
    */

    namedWindow("Result", 1);

	createTrackbar("Alpha", "Result", &alpha_, 180);
	createTrackbar("Beta", "Result", &beta_, 180);
	createTrackbar("Gamma", "Result", &gamma_, 180);
	createTrackbar("f", "Result", &f_, 2000);
	createTrackbar("Distance", "Result", &dist_, 2000);




    cout << "Capture is opened" << endl;
    for(;;)
    {
        capture >> image;
        //stampo il tipo di immagine
        if(flag == 0){
            string ty = "CV_" + type2str( image.type() );
            cout << "tipo matrice :" << ty.c_str() <<endl;
            flag = 1;
        }
        resize(image, image,Size(FRAMEWIDTH, FRAMEHEIGHT));



		          
        birdsEyeView(image, output);
        //CUDA_birdsEyeView(image, output);
        
        //per la visualizzazione 
        if(output.empty())
            break;
        //drawText(image);
        imshow("Result", output);
        if(waitKey(10) >= 0)
            break;
    }
    
    
    return 0;
}