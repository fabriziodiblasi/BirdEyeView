#include "hip/hip_runtime.h"
//#include "/home/fabrizio/Documents/Progetto_HPC/lib/utilities.h"
#include "../lib/utilities.h"

// namespaces
using namespace std;
using namespace cv;
#define PI 3.1415926


// int frameWidth = 640;
// int frameHeight = 480;
#define FRAMEWIDTH  640
#define FRAMEHEIGHT 480


// ---- GLOBAL VAR ----
int alpha_ = 90, beta_ = 90, gamma_ = 90;
int f_ = 500, dist_ = 500;



string type2str(int type) {
	string r;
  
	uchar depth = type & CV_MAT_DEPTH_MASK;
	uchar chans = 1 + (type >> CV_CN_SHIFT);
  
	switch ( depth ) {
	  case CV_8U:  r = "8U"; break;
	  case CV_8S:  r = "8S"; break;
	  case CV_16U: r = "16U"; break;
	  case CV_16S: r = "16S"; break;
	  case CV_32S: r = "32S"; break;
	  case CV_32F: r = "32F"; break;
	  case CV_64F: r = "64F"; break;
	  default:     r = "User"; break;
	}
  
	r += "C";
	r += (chans+'0');
  
	return r;
}


__global__ void rotation_multiply_kernel(float *d_RX,float *d_RY,float *d_R, int N){
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += d_RX[ROW * N + i] * d_RY[i * N + COL];
        }
    }
    d_R[ROW * N + COL] = tmpSum;

}


void birdsEyeView(const Mat &input, Mat &output){
    double focalLength, dist, alpha, beta, gamma; 

    alpha =((double)alpha_ -90) * PI/180;
    beta =((double)beta_ -90) * PI/180;
    gamma =((double)gamma_ -90) * PI/180;
    focalLength = (double)f_;
    dist = (double)dist_;

    Size input_size = input.size();
    double w = (double)input_size.width, h = (double)input_size.height;


    // Projecion matrix 2D -> 3D
    
    Mat A1 = (Mat_<float>(4, 3)<< 
        1, 0, -w/2,
        0, 1, -h/2,
        0, 0, 0,
        0, 0, 1 );
    
    
    // Rotation matrices Rx, Ry, Rz

    Mat RX = (Mat_<float>(4, 4) << 
        1, 0, 0, 0,
        0, cos(alpha), -sin(alpha), 0,
        0, sin(alpha), cos(alpha), 0,
        0, 0, 0, 1 );

    Mat RY = (Mat_<float>(4, 4) << 
        cos(beta), 0, -sin(beta), 0,
        0, 1, 0, 0,
        sin(beta), 0, cos(beta), 0,
        0, 0, 0, 1	);

    Mat RZ = (Mat_<float>(4, 4) << 
        cos(gamma), -sin(gamma), 0, 0,
        sin(gamma), cos(gamma), 0, 0,
        0, 0, 1, 0,
        0, 0, 0, 1	);

    // R - rotation matrix
    Mat R = RX * RY * RZ;

    cout<< " R : "<< R << endl;


    // T - translation matrix
    Mat T = (Mat_<float>(4, 4) << 
        1, 0, 0, 0,  
        0, 1, 0, 0,  
        0, 0, 1, dist,  
        0, 0, 0, 1); 
    
    // K - intrinsic matrix 
    Mat K = (Mat_<float>(3, 4) << 
        focalLength, 0, w/2, 0,
        0, focalLength, h/2, 0,
        0, 0, 1, 0
        ); 


    Mat transformationMat = K * (T * (R * A1));
    //cout<< "transformationMat.rows : " << transformationMat.rows << "\ttransformationMat.cols : " << transformationMat.cols << endl;
    //cout << "tipo matrice di transformazione : "<< "CV_" + type2str( transformationMat.type()) << endl;

    warpPerspective(input, output, transformationMat, input_size, INTER_CUBIC | WARP_INVERSE_MAP);

    return;



}





void CUDA_birdsEyeView(const Mat &input, Mat &output){

    double focalLength, dist, alpha, beta, gamma; 

    alpha =((double)alpha_ -90) * PI/180;
    beta =((double)beta_ -90) * PI/180;
    gamma =((double)gamma_ -90) * PI/180;
    focalLength = (double)f_;
    dist = (double)dist_;

    Size input_size = input.size();
    double w = (double)input_size.width, h = (double)input_size.height;
    /*
    compito :
    parallelizzare la funzione birdsEyeView
    aggiungere il file che fa il prodotto tra matrici in cuda
    */

    return;

}




int main(int argc, char const *argv[]) {
	
	if(argc > 2) {
      cerr << "Usage: " << argv[0] << " ' /path/to/video/ '  or nothing " << endl;
      cout << "Exiting...." << endl;
      return -1;
    }
    int flag=0;
    Mat image,output;
    

    VideoCapture capture;

    if (argc == 1){
        capture.open(0);
    }else{
        string filename = argv[1];
        capture.open(filename);
    }

    if(!capture.isOpened()) throw "Error reading video";

    

    /*
        definisco i parametri e le trackbar
    */

    namedWindow("Result", 1);

	createTrackbar("Alpha", "Result", &alpha_, 180);
	createTrackbar("Beta", "Result", &beta_, 180);
	createTrackbar("Gamma", "Result", &gamma_, 180);
	createTrackbar("f", "Result", &f_, 2000);
	createTrackbar("Distance", "Result", &dist_, 2000);




    cout << "Capture is opened" << endl;
    for(;;)
    {
        capture >> image;
        //stampo il tipo di immagine
        if(flag == 0){
            string ty = "CV_" + type2str( image.type() );
            cout << "tipo matrice :" << ty.c_str() <<endl;
            flag = 1;
        }
        resize(image, image,Size(FRAMEWIDTH, FRAMEHEIGHT));



		          
        birdsEyeView(image, output);
        
        
        //per la visualizzazione 
        if(output.empty())
            break;
        //drawText(image);
        imshow("Result", output);
        if(waitKey(10) >= 0)
            break;
    }
    
    
    return 0;
}